#include <cv_bridge/cv_bridge.h>
#include <image_transport/image_transport.hpp>
#include <opencv2/opencv.hpp>
#include <rclcpp/rclcpp.hpp>
#include <sensor_msgs/msg/image.hpp>
#include <string>
#include <sstream>
#include <fstream>
#include <filesystem>
#include <nlohmann/json.hpp>

#include "game_piece_detection/ModelInference.h"
#include "game_piece_detection/yolo_detection.h"

#include "vision_utils/publisher_queue.hpp"

#include "ament_index_cpp/get_package_share_directory.hpp"

namespace fs = std::filesystem;
using json = nlohmann::json;

class GamePieceDetector : public rclcpp::Node {
public:
GamePieceDetector()
      : Node("game_piece_detector") {

    // Decare parameters
    this->declare_parameter<std::string>("topic_name", "camera/image_raw");
    std::string topic_name = this->get_parameter("topic_name").as_string();

    this->declare_parameter<std::string>("camera_serial", "N/A");
    camera_serial_ = this->get_parameter("camera_serial").as_string();

    this->declare_parameter<std::string>("engine_file", "N/A");
    engine_file_ = this->get_parameter("engine_file").as_string();

    this->declare_parameter<std::string>("publish_to_topic",
                                         "game_piece_detector/images");
    publish_to_topic_ = this->get_parameter("publish_to_topic").as_string();

    subscriber_ = this->create_subscription<sensor_msgs::msg::Image>(
        topic_name, 1,
        std::bind(&GamePieceDetector::imageCallback, this,
                  std::placeholders::_1));

    // Game piece detector setup
    get_extrinsic_params();

    auto start = std::chrono::high_resolution_clock::now();
    // TODO: Initalize ModelInference object with engine file.
    RCLCPP_INFO(this->get_logger(),
                "ModelInference Initialization not implemented yet!");
    
    auto end = std::chrono::high_resolution_clock::now();
    auto processing_time =
        std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
            .count();
  }

  void init() {
    // The object needs to be constructed before using shared_from_this, thus
    // it is broken off into another method.
    it_ = std::make_shared<image_transport::ImageTransport>(shared_from_this());
    publisher_ = it_->advertise(publish_to_topic_, 10);
    image_pub_queue_ = std::make_shared<
        PublisherQueue<sensor_msgs::msg::Image, image_transport::Publisher>>(
        publisher_, 2);

    RCLCPP_INFO(this->get_logger(), "Publishing on topic: %s",
                publish_to_topic_.c_str());
  }

  ~GamePieceDetector() {
    image_pub_queue_->stop();
  }

private:

/**
   * @brief Load extrinsic parameters (rotation and offset) for this camera from
   * the system config file.
   *
   * Reads the rotation (3x3) and offset (3x1) from the extrinsics record for
   * the camera position, and stores them in OpenCV cv::Mat (rotation) and
   * cv::Vec3d (offset).
   */
  void get_extrinsic_params() {
    // Locate the system config file.
    fs::path config_path =
        ament_index_cpp::get_package_share_directory("vision_config_data");
    fs::path config_file = config_path / "data" / "system_config.json";

    if (!std::filesystem::exists(config_file)) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find system config file at path: %s",
                   config_file.c_str());
      return;
    }

    // Load the parameters from the file.
    std::ifstream f(config_file);
    json data = json::parse(f);

    if (!data.contains("camera_mounted_positions")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"camera_mounted_positions\" in system "
                   "config file");
      return;
    }
    if (!data.contains("extrinsics")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"extrinsics\" in system config file");
      return;
    }

    std::string camera_position = "N/A";
    if (data["camera_mounted_positions"].contains(camera_serial_)) {
      camera_position =
          data["camera_mounted_positions"][camera_serial_].get<std::string>();
    } else {
      RCLCPP_ERROR(
          this->get_logger(),
          "Unable to find camera serial %s in camera_mounted_positions in "
          "system config file",
          camera_serial_.c_str());
      return;
    }
    if (!data["extrinsics"].contains(camera_position)) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find camera position %s in extrinsics in system "
                   "config file",
                   camera_position.c_str());
      return;
    }

    RCLCPP_INFO(this->get_logger(),
                "Found camera position %s for camera serial %s",
                camera_position.c_str(), camera_serial_.c_str());

    // Read rotation and offset
    const auto &extr = data["extrinsics"][camera_position];
    if (!extr.contains("rotation") || !extr.contains("offset")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Extrinsics for %s missing 'rotation' or 'offset' field",
                   camera_position.c_str());
      return;
    }
    // Read rotation (should be 3x3 array)
    cv::Mat rotation(3, 3, CV_64F);
    for (int i = 0; i < 3; ++i) {
      for (int j = 0; j < 3; ++j) {
        rotation.at<double>(i, j) = extr["rotation"][i][j];
      }
    }
    // Read offset (should be 3x1 array)
    cv::Vec3d offset;
    for (int i = 0; i < 3; ++i) {
      offset[i] = extr["offset"][i];
    }
    extrinsic_rotation_ = rotation;
    extrinsic_offset_ = cv::Mat(offset);

    RCLCPP_INFO(
        this->get_logger(),
        "Loaded extrinsics for %s: "
        "rotation=[[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]], offset=[%f,%f,%f]",
        camera_position.c_str(), rotation.at<double>(0, 0),
        rotation.at<double>(0, 1), rotation.at<double>(0, 2),
        rotation.at<double>(1, 0), rotation.at<double>(1, 1),
        rotation.at<double>(1, 2), rotation.at<double>(2, 0),
        rotation.at<double>(2, 1), rotation.at<double>(2, 2), offset[0],
        offset[1], offset[2]);
  }

  void get_network_tables_params() {
    // Locate the system config file.
    fs::path config_path =
        ament_index_cpp::get_package_share_directory("vision_config_data");
    fs::path config_file = config_path / "data" / "system_config.json";

    if (!std::filesystem::exists(config_file)) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find system config file at path: %s",
                   config_file.c_str());
    }

    // Load the parameters from the file.
    std::ifstream f(config_file);
    json data = json::parse(f);

    if (!data.contains("network_tables_config")) {
      RCLCPP_ERROR(
          this->get_logger(),
          "Unable to find key \"network_tables_config\" in system config file");
    }
    if (!data["network_tables_config"].contains("table_address")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"table_address\" in "
                   "\"network_tables_config\" record in system config file");
    }
    if (!data["network_tables_config"].contains("table_name")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"table_name\" in "
                   "\"network_tables_config\" record in system config file");
    }

    data["network_tables_config"].at("table_address").get_to(table_address_);
    data["network_tables_config"].at("table_name").get_to(table_name_);

    RCLCPP_INFO(
        this->get_logger(),
        "Loaded network tables parameters table_address: %s, table_name: %s",
        table_address_.c_str(), table_name_.c_str());

    return;
  }

  void imageCallback(const sensor_msgs::msg::Image::SharedPtr msg) {

    cv::Mat yuyv_img;

    auto start = std::chrono::high_resolution_clock::now();
    cv::Mat bgr_img = cv_bridge::toCvCopy(msg, "bgr8")->image;

    //TODO: Run the inference and publish the output to a message.

    
    // Publish the message to the viewer
    // auto outgoing_msg =
    //     cv_bridge::CvImage(std_msgs::msg::Header(), "bgr8", bgr_img)
    //         .toImageMsg();
    // outgoing_msg->header.stamp = this->now();
    // outgoing_msg->header.frame_id = "apriltag_detections";
    // image_pub_queue_->enqueue(outgoing_msg);

  }

  rclcpp::Subscription<sensor_msgs::msg::Image>::SharedPtr subscriber_;

  std::shared_ptr<image_transport::ImageTransport> it_;

  std::shared_ptr<
      PublisherQueue<sensor_msgs::msg::Image, image_transport::Publisher>>
      image_pub_queue_;
  image_transport::Publisher publisher_;
  std::string publish_to_topic_;
  std::string camera_serial_;
  std::string engine_file_;
};

int main(int argc, char *argv[]) {
  rclcpp::init(argc, argv);
  auto node = std::make_shared<GamePieceDetector>();
  node->init();
  rclcpp::spin(node);
  rclcpp::shutdown();
  return 0;
}