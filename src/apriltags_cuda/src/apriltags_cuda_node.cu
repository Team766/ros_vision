#include <cv_bridge/cv_bridge.h>

#include <filesystem>
#include <fstream>
#include <image_transport/image_transport.hpp>
#include <nlohmann/json.hpp>
#include <opencv2/opencv.hpp>
#include <rclcpp/rclcpp.hpp>
#include <sensor_msgs/msg/image.hpp>
#include <sstream>
#include <string>
#include <vector>


#include "DoubleArraySender.h"
#include "ament_index_cpp/get_package_share_directory.hpp"
#include "apriltag_gpu.h"
#include "apriltag_utils.h"
#include "vision_utils/publisher_queue.hpp"

extern "C" {
#include "apriltag.h"
#include "apriltag_pose.h"
#include "common/zarray.h"
}

#define TAGSIZE 0.1651  // tag size, in meters

namespace fs = std::filesystem;
using json = nlohmann::json;

class ApriltagsDetector : public rclcpp::Node {
 public:
  ApriltagsDetector()
      : Node("apriltags_detector"),
        tag_family_(nullptr),
        tag_detector_(nullptr) {
    // Decare parameters
    this->declare_parameter<std::string>("topic_name", "camera/image_raw");
    std::string topic_name = this->get_parameter("topic_name").as_string();

    this->declare_parameter<std::string>("camera_serial", "N/A");
    camera_serial_ = this->get_parameter("camera_serial").as_string();

    this->declare_parameter<std::string>("publish_to_topic",
                                         "apriltags/images");
    publish_to_topic_ = this->get_parameter("publish_to_topic").as_string();

    subscriber_ = this->create_subscription<sensor_msgs::msg::Image>(
        topic_name, 1,
        std::bind(&ApriltagsDetector::imageCallback, this,
                  std::placeholders::_1));

    // Apriltag detector setup
    setup_tag_family(&tag_family_, tag_family_name_);
    tag_detector_ = apriltag_detector_create();
    apriltag_detector_add_family(tag_detector_, tag_family_);

    tag_detector_->quad_decimate = 2.0;
    tag_detector_->quad_sigma = 0.0;
    tag_detector_->nthreads = 1;
    tag_detector_->debug = false;
    tag_detector_->refine_edges = true;
    tag_detector_->wp = workerpool_create(4);

    // TODO: read these from a file or a topic.
    frc971::apriltag::CameraMatrix cam;
    frc971::apriltag::DistCoeffs dist;

    get_camera_calibration_data(&cam, &dist);
    get_extrinsic_params();
    get_network_tables_params();

    // Create the tag sender for sending tag data to the network tables.
    // TODO: may need to use the camera idx rather than the serial number.
    tag_sender_ = std::make_shared<DoubleArraySender>(
        camera_serial_, table_address_, table_name_);

    int frame_width = 1280;
    int frame_height = 800;

    auto start = std::chrono::high_resolution_clock::now();
    detector_ = new frc971::apriltag::GpuDetector(frame_width, frame_height,
                                                  tag_detector_, cam, dist);
    auto end = std::chrono::high_resolution_clock::now();
    auto processing_time =
        std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
            .count();

    info_.tagsize = TAGSIZE;
    info_.fx = cam.fx;
    info_.fy = cam.fy;
    info_.cx = cam.cx;
    info_.cy = cam.cy;

    RCLCPP_INFO(this->get_logger(),
                "GPU Apriltag Detector created, took %ld ms", processing_time);
  }

  void init() {
    // The object needs to be constructed before using shared_from_this, thus
    // it is broken off into another method.
    it_ = std::make_shared<image_transport::ImageTransport>(shared_from_this());
    publisher_ = it_->advertise(publish_to_topic_, 10);
    image_pub_queue_ = std::make_shared<
        PublisherQueue<sensor_msgs::msg::Image, image_transport::Publisher>>(
        publisher_, 2);

    RCLCPP_INFO(this->get_logger(), "Publishing on topic: %s",
                publish_to_topic_.c_str());
  }

  ~ApriltagsDetector() {
    apriltag_detector_destroy(tag_detector_);
    teardown_tag_family(&tag_family_, tag_family_name_);
    delete detector_;
    image_pub_queue_->stop();
  }

 private:
  void get_extrinsic_params() {
    // TODO: Implement a method to parse the extrinsics parameters from the
    //  config files.
    RCLCPP_INFO(this->get_logger(),
                "Extrinsics parameters not yet implemented!");

    return;
  }

  void get_network_tables_params() {
    // Locate the system config file.
    fs::path config_path =
        ament_index_cpp::get_package_share_directory("vision_config_data");
    fs::path config_file = config_path / "data" / "system_config.json";

    if (!std::filesystem::exists(config_file)) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find system config file at path: %s",
                   config_file.c_str());
    }

    // Load the parameters from the file.
    std::ifstream f(config_file);
    json data = json::parse(f);

    if (!data.contains("network_tables_config")) {
      RCLCPP_ERROR(
          this->get_logger(),
          "Unable to find key \"network_tables_config\" in system config file");
    }
    if (!data["network_tables_config"].contains("table_address")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"table_address\" in "
                   "\"network_tables_config\" record in system config file");
    }
    if (!data["network_tables_config"].contains("table_name")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"table_name\" in "
                   "\"network_tables_config\" record in system config file");
    }

    data["network_tables_config"].at("table_address").get_to(table_address_);
    data["network_tables_config"].at("table_name").get_to(table_name_);

    RCLCPP_INFO(
        this->get_logger(),
        "Loaded network tables parameters table_address: %s, table_name: %s",
        table_address_.c_str(), table_name_.c_str());

    return;
  }

  void get_camera_calibration_data(frc971::apriltag::CameraMatrix *cam,
                                   frc971::apriltag::DistCoeffs *dist) {
    // Locate the calibration file for the camera serial id that we are
    // processing.
    fs::path config_path =
        ament_index_cpp::get_package_share_directory("vision_config_data");
    std::ostringstream oss;
    oss << "calibrationmatrix_" << camera_serial_ << ".json";
    fs::path cal_file = config_path / "data" / "calibration" / oss.str();

    if (!std::filesystem::exists(cal_file)) {
      RCLCPP_ERROR(
          this->get_logger(),
          "Unable to find calibration parameters for camera %s at path: %s",
          camera_serial_.c_str(), cal_file.c_str());
    }

    // Load the parameters from the file.
    std::ifstream f(cal_file);
    json data = json::parse(f);

    if (!data.contains("matrix")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"matrix\" in calibration file");
    }
    if (!data.contains("disto")) {
      RCLCPP_ERROR(this->get_logger(),
                   "Unable to find key \"disto\" in calibration file");
    }

    RCLCPP_INFO(this->get_logger(),
                "Loading camera calibration params from: %s", cal_file.c_str());

    // Setup Camera Matrix
    // Intrinsic Matrices are explained here:
    // https://docs.opencv.org/4.x/d9/d0c/group__calib3d.html
    cam->fx = data["matrix"][0][0];
    cam->fy = data["matrix"][1][1];
    cam->cx = data["matrix"][0][2];
    cam->cy = data["matrix"][1][2];

    // Setup Distortion Coefficients
    // OpenCV writes them out in the order specified here:
    // https://docs.opencv.org/4.x/d9/d0c/group__calib3d.html
    dist->k1 = data["disto"][0][0];
    dist->k2 = data["disto"][0][1];
    dist->p1 = data["disto"][0][2];
    dist->p2 = data["disto"][0][3];
    dist->k3 = data["disto"][0][4];

    RCLCPP_INFO(
        this->get_logger(),
        "Loaded camera matrix parameters: fx: %f, fy: %f, cx: %f, cy: %f",
        cam->fx, cam->fy, cam->cx, cam->cy);
    RCLCPP_INFO(
        this->get_logger(),
        "Loaded distortion coeffs k1: %f, k2: %f, p1: %f, p2: %f, k3: %f",
        dist->k1, dist->k2, dist->p1, dist->p2, dist->k3);
  }

  void imageCallback(const sensor_msgs::msg::Image::SharedPtr msg) {
    cv::Mat yuyv_img;

    rclcpp::Time image_capture_time = msg->header.stamp;

    auto start = std::chrono::high_resolution_clock::now();
    auto cv_ptr = cv_bridge::toCvCopy(msg, "bgr8");  // Use smart pointer
    cv::Mat bgr_img = cv_ptr->image;

    cv::cvtColor(bgr_img, yuyv_img, cv::COLOR_BGR2YUV_YUYV);

    auto detstart = std::chrono::high_resolution_clock::now();
    detector_->Detect(yuyv_img.data);
    auto detend = std::chrono::high_resolution_clock::now();
    auto det_processing_time =
        std::chrono::duration_cast<std::chrono::milliseconds>(detend - detstart)
            .count();

    const zarray_t *detections = detector_->Detections();
    draw_detection_outlines(bgr_img, const_cast<zarray_t *>(detections));

    std::vector<double> networktables_pose_data = {};
    if (zarray_size(detections) > 0) {
      for (int i = 0; i < zarray_size(detections); i++) {
        apriltag_detection_t *det;
        zarray_get(const_cast<zarray_t *>(detections), i, &det);

        // Setup the detection info struct for use down below.
        info_.det = det;

        apriltag_pose_t pose;
        double err = estimate_tag_pose(&info_, &pose);
        cv::Vec3d aprilTagInCameraFrame(pose.t->data[0], pose.t->data[1],
                                        pose.t->data[2]);

        RCLCPP_DEBUG(this->get_logger(),
                     "Tag id: %d, x: %.6f, y: %.6f, z: %.6f, err: %.6f",
                     det->id, pose.t->data[0], pose.t->data[1], pose.t->data[2],
                     err);
        networktables_pose_data.push_back(image_capture_time.seconds());
        networktables_pose_data.push_back(det->id * 1.0);

        networktables_pose_data.push_back(aprilTagInCameraFrame[0]);
        networktables_pose_data.push_back(aprilTagInCameraFrame[1]);
        networktables_pose_data.push_back(aprilTagInCameraFrame[2]);
      }
      detector_->ReinitializeDetections();
    }

    // Send the pose data to the network tables.
    tag_sender_->sendValue(networktables_pose_data);

    // Publish the message to the viewer
    auto outgoing_msg =
        cv_bridge::CvImage(std_msgs::msg::Header(), "bgr8", bgr_img)
            .toImageMsg();
    outgoing_msg->header.stamp = this->now();
    outgoing_msg->header.frame_id = "apriltag_detections";
    image_pub_queue_->enqueue(outgoing_msg);

    auto end = std::chrono::high_resolution_clock::now();
    auto processing_time =
        std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
            .count();

    RCLCPP_DEBUG(this->get_logger(), "Total Time: %ld ms, Det Time: %ld ms",
                 processing_time, det_processing_time);
  }

  rclcpp::Subscription<sensor_msgs::msg::Image>::SharedPtr subscriber_;

  std::shared_ptr<image_transport::ImageTransport> it_;

  std::shared_ptr<
      PublisherQueue<sensor_msgs::msg::Image, image_transport::Publisher>>
      image_pub_queue_;
  image_transport::Publisher publisher_;
  std::string publish_to_topic_;
  std::string camera_serial_;
  std::string table_address_;
  std::string table_name_;

  std::shared_ptr<DoubleArraySender> tag_sender_;

  apriltag_family_t *tag_family_;
  apriltag_detector_t *tag_detector_;
  apriltag_detection_info_t info_;
  frc971::apriltag::GpuDetector *detector_;
  const char *tag_family_name_ = "tag36h11";
};

int main(int argc, char *argv[]) {
  rclcpp::init(argc, argv);
  auto node = std::make_shared<ApriltagsDetector>();
  node->init();
  rclcpp::spin(node);
  rclcpp::shutdown();
  return 0;
}